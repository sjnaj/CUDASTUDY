#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "parallelprogrammingbook/include/hpc_helpers.hpp"
#include "common/cpu_bitmap.h"
#include "common/cpu_anim.h"
#include "common/book.h"
#define DIM 1024
#define MAX_TEMP 0.8f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

/*heating cells remains at a constant temperature*/
__global__ void copy_const_kernel(float *iptr, float *const cptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if (cptr[offset] != 0)
    {
        iptr[offset] = cptr[offset];
    }
}

__global__ void blend_kernel(float *outSrc, float *const inSrc)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)
        left++;
    if (x == DIM - 1)
        right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)
        top += DIM;
    if (y == DIM - 1)
        bottom -= DIM;

    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - 4 * inSrc[offset]);
}

struct DataBlock
{
    unsigned char *output_bitmap;
    float *dev_inSrc;
    float *dev_outSrc;
    float *dev_constSrc;
    CPUAnimBitmap *bitmap;
    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

void anim_gpu(DataBlock *d, int ticks)
{
    hipEventRecord(d->start, 0);
    CUERR
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;

    for (int i = 0; i < 90; i++)
    {
        copy_const_kernel<<<blocks, threads>>>(d->dev_inSrc, d->dev_constSrc);
        blend_kernel<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
        swap(d->dev_inSrc, d->dev_outSrc);
    }

    float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);

    hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
    CUERR
    hipEventRecord(d->stop, 0);
    CUERR
    hipEventSynchronize(d->stop);
    CUERR
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, d->start, d->stop);
    CUERR
    d->totalTime += elapsedTime;

    d->frames++;
    printf("average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d)
{
    hipFree(d->dev_inSrc);
    CUERR
    hipFree(d->dev_outSrc);
    CUERR
    hipFree(d->dev_constSrc);
    CUERR

    hipEventDestroy(d->start);
    CUERR
    hipEventDestroy(d->stop);
    CUERR
}

int main(void)
{

    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;

    hipEventCreate(&data.start);
    CUERR
    hipEventCreate(&data.stop);
    CUERR
    hipMalloc((void **)&data.output_bitmap, bitmap.image_size());
    CUERR
    hipMalloc((void **)&data.dev_inSrc, bitmap.image_size());
    CUERR
    hipMalloc((void **)&data.dev_outSrc, bitmap.image_size());
    CUERR
    hipMalloc((void **)&data.dev_constSrc, bitmap.image_size());
    CUERR
    float *temp;
    hipHostMalloc((void **)&temp, bitmap.image_size());
    CUERR

    // 初始化内部热点
    for (int i = 0; i < DIM * DIM; i++)
    {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
        {
            temp[i] = MAX_TEMP;
        }
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++)
    {
        for (int x = 400; x < 500; x++)
        {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }

    hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR
    // 初始化外部影响
    for (int y = 800; y < DIM; y++)
    {
        for (int x = 0; x < 200; x++)
        {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR

    hipHostFree(temp);
    CUERR

    bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void *))anim_exit);
}
