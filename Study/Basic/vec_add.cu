#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/book.h"
#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) // 一个线程处理对应一列，不限长度
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<128, 128>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    bool sucess = true;

    for (int i = 0; i < N; i++)
    {
        if ((a[i] + b[i]) != c[i])
        {
            printf("error\n");
            sucess = false;
        }
    }
    if (sucess)
    {
        printf("okk\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}