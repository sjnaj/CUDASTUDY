#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <algorithm>
#include <iostream>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_math_constants.h>
using namespace std;

// source from https://github.com/Oneflow-Inc/oneflow/blob/master/oneflow/core/cuda/softmax.cuh

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);}

constexpr int kWarpSize = 32;

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return max(a, b); }
};

// 每个 Warp 处理一行或两行元素，每行的Reduce操作 需要做 Warp 内的 Reduce 操作，
// 我们实现 WarpAllReduce 来完成 Warp 内各线程间的求 Global Max 和 Global Sum 操作，
// WarpAllReduce 是利用Warp级别原语 __shfl_xor_sync 实现的，代码如下。
template<template<typename> class ReductionOp, typename T, int thread_group_width = kWarpSize>
__inline__ __device__ T WarpAllReduce(T val) {
  for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
    val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

// BlockReduce 使用 cub 进行实现
template<template<typename> class ReductionOp, typename T, int block_size>
__inline__ __device__ T BlockAllReduce(T val) {
  typedef hipcub::BlockReduce<T, block_size> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T result_broadcast;
  T result = BlockReduce(temp_storage).Reduce(val, ReductionOp<T>());
  if (threadIdx.x == 0) { result_broadcast = result; }
  __syncthreads();
  return result_broadcast;
}

// 定义各种数据类型下面的 inf 值
template<typename T>
__inline__ __device__ T Inf();

template<>
__inline__ __device__ float Inf<float>() {
  return HIP_INF_F;
}

template<>
__inline__ __device__ double Inf<double>() {
  return HIP_INF;
}

// 定义 exp 函数，OF_SOFTMAX_USE_FAST_MATH 这个宏表示开启 FAST_MATH 的 exp
template<typename T>
__inline__ __device__ T Exp(T x);

template<>
__inline__ __device__ float Exp<float>(float x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __expf(x);
#else
  return exp(x);
#endif
}

template<>
__inline__ __device__ double Exp<double>(double x) {
  return exp(x);
}

// 定义 div 函数，OF_SOFTMAX_USE_FAST_MATH 这个宏表示开启 FAST_MATH 的 div
template<typename T>
__inline__ __device__ T Div(T a, T b);

template<>
__inline__ __device__ float Div<float>(float a, float b) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __fdividef(a, b);
#else
  return a / b;
#endif
}

template<>
__inline__ __device__ double Div<double>(double a, double b) {
  return a / b;
}

// 定义 log 函数，OF_SOFTMAX_USE_FAST_MATH 这个宏表示开启 FAST_MATH 的 log
template<typename T>
__inline__ __device__ T Log(T x);

template<>
__inline__ __device__ float Log<float>(float x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __logf(x);
#else
  return log(x);
#endif
}
template<>
__inline__ __device__ double Log<double>(double x) {
  return log(x);
}

// 对于 cuda kernel 来说，启动多少个线程块（grid_size）来做计算？
// 具体可以参考俊丞大佬这篇 [如何设置CUDA Kernel中的grid_size和block_size？ ](https://mp.weixin.qq.com/s/1_ao9xM6Qk3JaavptChXew)
inline hipError_t GetNumBlocks(int64_t block_size, int64_t max_blocks, int64_t waves,
                                int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks =
      std::max<int>(1, std::min<int64_t>(max_blocks, sm_count * tpm / block_size * waves));
  return hipSuccess;
}

// 定义默认的计算类型，一般的数据类型进行 softmax 计算时计算类型就是自己，
// 而对于 half（或者bfp16） 来说我们往往需要将计算类型 fallback 到 float 来避免精度损失
template<typename T>
struct DefaultComputeType {
  using type = T;
};

template<>
struct DefaultComputeType<half> {
  using type = float;
};

#if CUDA_VERSION >= 11000
template<>
struct DefaultComputeType<hip_bfloat16> {
  using type = float;
};
#endif  // CUDA_VERSION >= 11000

// GetPackType 结构体中使用了 std::aligned_storage 先声明了一个内存对齐的数据类型 type ，
// 注意这个 type 的内存长度为 pack_size * sizeof(T) 。然后这里的 T 是我们需要进行 Pack 
// 的数据类型，而 pack_size 则表示我们需要 Pack 的元素个数。接下来我们看到 Pack 联合体中
// 声明了 storage 和 elem 两个数组，它们共用同一段对齐的内存。然后 Pack 联合体的入口有一个
// 检查: static_assert(sizeof(PackType<T, pack_size>) == sizeof(T) * pack_size, ""); 
// 这是用来判断我们之前声明的 type 的内存长度是否符合预期。
template<typename T, int N>
struct GetPackType {
  using type = typename std::aligned_storage<N * sizeof(T), N * sizeof(T)>::type;
};

template<typename T, int N>
using PackType = typename GetPackType<T, N>::type;

template<typename T, int N>
union Pack {
  static_assert(sizeof(PackType<T, N>) == sizeof(T) * N, "");
  __device__ Pack() {
    // do nothing
  }
  PackType<T, N> storage;
  T elem[N];
};

// 下面分别定义了两个代表输入输出的数据结构
template<typename SRC, typename DST>
struct DirectLoad {
  DirectLoad(const SRC* src, int64_t row_size) : src(src), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    Pack<SRC, N> pack;
    const int64_t offset = (row * row_size + col) / N;
    pack.storage = *(reinterpret_cast<const PackType<SRC, N>*>(src) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }
  const SRC* src;
  int64_t row_size;
};

template<typename SRC, typename DST>
struct DirectStore {
  DirectStore(DST* dst, int64_t row_size) : dst(dst), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    Pack<DST, N> pack;
    const int64_t offset = (row * row_size + col) / N;
#pragma unroll
    for (int i = 0; i < N; ++i) { pack.elem[i] = static_cast<DST>(src[i]); }
    *(reinterpret_cast<PackType<DST, N>*>(dst) + offset) = pack.storage;
  }
  DST* dst;
  int64_t row_size;
};

// 
enum class Algorithm {
  kSoftmax = 0,
  kLogSoftmax = 1,
};

// 使用 `load.template load<pack_size>(ptr, row_id, col_id);`
// 和`store.template store<pack_size>(ptr, row_id, col_id);` 进行读取和写入
// 使用LOAD和STORE有两个好处：1、可以在CUDA Kernel中只关心计算类型ComputeType，而不用关心具体的数据类型T。
// 2、只需要加几行代码就可以快速支持Softmax和其他Kernel Fuse，减少带宽需求，提升整体性能。
// 普通的SoftmaxKernel直接使用DirectLoad和DirectStore，FusedSoftmaxKernel如FusedScaleSoftmaxDropoutKernel
// 只需要定义一个ScaleLoad结构和一个DropoutStore结构用于对输入x做Scale预处理和对输出y做Dropout后处理。
// ComputeType代表计算类型。pack_size代表向量化访存操作的pack元素的个数，我们将几个元素pack起来读写，提升带宽利用率。
// cols_per_thread代表每个线程处理的元素个数。thread_group_width代表处理元素的线程组的宽度，
// 当cols > pack_size * warp_size时，thread_group_width就是warp_size，即32。
// 当cols < pack_size * warp_size时，就根据cols大小用1/2个warp或1/4个warp来处理每行的元素。
// 采用更小的thread_group_width后，WarpAllReduce需要执行的轮次也相应减少。
// rows_per_access代表每个线程组一次处理的行数，当cols较小，thread_group_width不是warp_size 32时，
// 若rows能被2整除，我们就让每个线程处理2行来增加指令并行度，从而提升性能。
// padding代表当前是否做了padding，若cols不是warp_size的整数倍，我们会把它padding到最近的整数倍处理。
// algorithm代表使用的算法，可选项有Algorithm::kSoftmax或Algorithm::kLogSoftmax。

// CUDA Kernel执行的主体循环逻辑如下，首先根据 num_cols信息算出每个线程要处理的cols_per_thread，
// 每个线程分配`rows_per_access * cols_per_thread`大小的寄存器，将输入x读到寄存器中，后续计算均从寄存器中读取。
template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, bool padding, Algorithm algorithm>
__global__ void SoftmaxWarpImpl(LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  // 我们需要保证每个线程处理的元素个数可以被 pack_size 整除
  static_assert(cols_per_thread % pack_size == 0, "");
  // 处理元素的线程组的宽度需要小于等于kWarpSize，并且需要被kWarpSize整除
  static_assert(thread_group_width <= kWarpSize, "");
  static_assert(kWarpSize % thread_group_width == 0, "");
  // 每个线程处理的 pack 后的元素个数
  constexpr int num_packs = cols_per_thread / pack_size;
  // 需要保证 cols <= 每个线程处理的元素个数 * 处理元素的线程组的宽度 ，因为这个地方是每个 Warp 处理一行或两行元素
  assert(cols <= cols_per_thread  * thread_group_width);
  // 开一块共享内存，行数为每个线程组一次处理的行数，列数为每个线程处理的元素个数
  ComputeType buf[rows_per_access][cols_per_thread];

  // int grid_dim_x;
  // dim3 block_dim(thread_group_width, thread_groups_per_block);
  // 从下面启动 SoftmaxWarpImpl 的参数来看，这里使用的是一维的 grid，二维的 block，
  // 并且 block 的长度为处理元素的线程组的宽度（warp_size），block 的宽度为每个 block 的线程组的个数
  // 注意启动 kernel 时每个 block 的总线程数是 128 ，如果 thread_group_width = 32
  // 那么 thread_groups_per_block = 128 / 32 = 4
  // 获取全局的线程组id
  const int global_thread_group_id = blockIdx.x * blockDim.y + threadIdx.y;
  // 获取全局线程组的数量
  const int num_global_thread_group = gridDim.x * blockDim.y;
  // lane id，表示当前 thread 在当前 lane 中的索引，注意 threadIdx.x 在这个 kernel 里面是不可能超越一个 warp (32) 的
  // 所以这里省掉了取模
  const int lane_id = threadIdx.x;
  // step 表示循环计数器大小为 全局线程组的数量 * 每个线程组一次处理的行数
  const int64_t step = num_global_thread_group * rows_per_access;
  // for 循环的开始为 row = 全局的线程组id * 每个线程组一次处理的行数，结束为总行数
  for (int64_t row = global_thread_group_id * rows_per_access; row < rows; row += step) {
    // 开辟一块共享内存记录当前线程组处理的每一行的最大值
    ComputeType thread_max[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      // 把 thread_max[row_id] 初始化为 -inf
      thread_max[row_id] = -Inf<ComputeType>();
      // 获取第 row_id 行的共享内存数据
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < num_packs; ++pack_id) {
        // pack的偏移量
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          load.template load<pack_size>(row_buf + pack_offset, row + row_id, col);
#pragma unroll
          for (int i = 0; i < pack_size; ++i) {
            thread_max[row_id] = max(thread_max[row_id], row_buf[pack_offset + i]);
          }
        } else {
#pragma unroll
          for (int i = 0; i < pack_size; ++i) { row_buf[pack_offset + i] = -Inf<ComputeType>(); }
        }
      }
    }
    // 开辟一块共享内存记录属于同一个warp的线程组的每一行的最大值，也就是需要进行一次warpReduce max
    ComputeType warp_max[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_max[row_id] = WarpAllReduce<MaxOp, ComputeType, thread_group_width>(thread_max[row_id]);
    }
    // 开辟一块共享内存记录当前线程组处理的每一行的sum
    ComputeType thread_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_sum[row_id] = 0;
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int i = 0; i < cols_per_thread; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          row_buf[i] = Exp(row_buf[i] - warp_max[row_id]);
          thread_sum[row_id] += row_buf[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          row_buf[i] -= warp_max[row_id];
          thread_sum[row_id] += Exp(row_buf[i]);
        } else {
          __trap(); // 内核的执行被中止并在主机程序中引发中断。
        }
      }
    }
    // 开辟一块共享内存记录属于同一个warp的线程组的每一行的sum (注意这里考虑了指数运算的安全性，
    // 实际上求的是sum{exp(x_i-max{x_i})})，也就是需要进行一次warpReduce sum
    ComputeType warp_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_sum[row_id] = WarpAllReduce<SumOp, ComputeType, thread_group_width>(thread_sum[row_id]);
    }
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int i = 0; i < cols_per_thread; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          row_buf[i] = Div(row_buf[i], warp_sum[row_id]);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          row_buf[i] -= Log(warp_sum[row_id]);
        } else {
          __trap();// 内核的执行被中止并在主机程序中引发中断。
        }
      }
#pragma unroll
      for (int i = 0; i < num_packs; ++i) {
        const int col = (i * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          store.template store<pack_size>(row_buf + i * pack_size, row + row_id, col);
        }
      }
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, bool padding, Algorithm algorithm>
inline hipError_t LaunchSoftmaxWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                         const int64_t rows, const int64_t cols) {
  // 每个 Block 里面设置 128 个线程
  constexpr int block_size = 128;
  // GPU 一次可以调度 SM 数量 * 每个 SM 最大 block 数个 block，
  // 因为每个 block 的计算量相等，所以所有 SM 应几乎同时完成这些 block 的计算，
  // 然后处理下一批，这其中的每一批被称之为一个 wave。
  // grid_size 设置为可以满足足够多的 wave，也就是这里定义的 waves
  constexpr int waves = 32;
  // block_size 需要整除处理元素的线程组的宽度
  static_assert(block_size % thread_group_width == 0, "");
  // 每个 block 的线程组的个数，如果这里 thread_group_width = 32，那么 thread_groups_per_block = 4
  constexpr int thread_groups_per_block = block_size / thread_group_width;
  dim3 block_dim(thread_group_width, thread_groups_per_block);
  // 根据数据的大小计算最多设置多少个 block ，就是数据的行数 / 每个线程组一次处理的行数
  const int64_t num_blocks =
      (rows / rows_per_access + thread_groups_per_block - 1) / thread_groups_per_block;
  // 根据上述的设置以及硬件本身的限制计算最终启动的 block 数
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, num_blocks, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread, thread_group_width,
                  rows_per_access, padding, algorithm>
      <<<grid_dim_x, block_dim, 0, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, Algorithm algorithm>
inline hipError_t DispatchSoftmaxWarpImplPadding(hipStream_t stream, LOAD load, STORE store,
                                                  const int64_t rows, const int64_t cols) {
  // 如果每个线程处理的元素个数 * 处理元素的线程组的宽度(warp_size)和cols相等，就不需要padding
  if (cols == cols_per_thread * thread_group_width) {
    return LaunchSoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread,
                                 thread_group_width, rows_per_access, false, algorithm>(
        stream, load, store, rows, cols);
  } else {
    return LaunchSoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread,
                                 thread_group_width, rows_per_access, true, algorithm>(
        stream, load, store, rows, cols);
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
typename std::enable_if<pack_size == 1, hipError_t>::type DispatchSoftmaxWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
// 这里是对比数据的列数(cols)和线程组的宽度*pack_size，如果满足条件就dispatch到对应的线程组宽度(warp_size)，
// 注意这里是一个线程处理 pack_size 个元素
#define DEFINE_ONE_ELIF(thread_group_width)                                                        \
  else if (cols <= (thread_group_width)*pack_size) {                                               \
    if (rows % 2 == 0) {                                                                           \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 2, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    } else {                                                                                       \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 1, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    }                                                                                              \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
// 如果上面的条件都不满足，那么直接一个线程处理col个元素，而不是pack_size个元素
#define DEFINE_ONE_ELIF(col)                                                                      \
  else if (cols <= (col)*kWarpSize) {                                                             \
    return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, col, kWarpSize, 1, \
                                          algorithm>(stream, load, store, rows, cols);            \
  }
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(3)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(5)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(7)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(9)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(11)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(13)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(15)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(17)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(19)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(21)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(23)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(25)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(27)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(29)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(31)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

// 同理，dispatch pack_size==2情况下的kernel
template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
typename std::enable_if<pack_size == 2, hipError_t>::type DispatchSoftmaxWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                        \
  else if (cols <= (thread_group_width)*pack_size) {                                               \
    if (rows % 2 == 0) {                                                                           \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 2, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    } else {                                                                                       \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 1, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    }                                                                                              \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                      \
  else if (cols <= (col)*kWarpSize) {                                                             \
    return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, col, kWarpSize, 1, \
                                          algorithm>(stream, load, store, rows, cols);            \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct DispatchSoftmaxWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols) {
    if (cols % 2 == 0) {
      return DispatchSoftmaxWarpImplCols<LOAD, STORE, ComputeType, 2, algorithm>(stream, load,
                                                                                 store, rows, cols);
    } else {
      return DispatchSoftmaxWarpImplCols<LOAD, STORE, ComputeType, 1, algorithm>(stream, load,
                                                                                 store, rows, cols);
    }
  }
};

// 每个Warp处理一行或两行元素时最原始的dispatch接口
template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t DispatchSoftmaxWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                           const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxWarpImplPackSize<LOAD, STORE, ComputeType, algorithm>()(stream, load, store,
                                                                                rows, cols);
}

// 一个 Block 处理一行元素， 利用 BlockAllReduce 完成 Warp 内各线程间的求 Global Max 和 Global Sum 操作。
// BlockAllReduce 是借助 Cub 的 BlockReduce 方法实现的。

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
__global__ void SoftmaxBlockSMemImpl(LOAD load, STORE store, const int64_t rows,
                                     const int64_t cols) {
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  auto* buf = reinterpret_cast<ComputeType*>(shared_buf);
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  // 一个 Block 处理一行元素
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    // 当前线程的最大值初始化为 -inf
    ComputeType thread_max = -Inf<ComputeType>();
    // 以向量化的方式加载一行数据，然后执行pack reduce操作
    // 这里的 pack reduce操作我在 https://zhuanlan.zhihu.com/p/596012674 最后一节也有介绍
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        buf[i * num_packs + pack_id] = pack[i];
        thread_max = max(thread_max, pack[i]);
      }
    }
    // 执行block reduce获取当前行（由一个 Block 进行处理）的最大值
    const ComputeType row_max = BlockAllReduce<MaxOp, ComputeType, block_size>(thread_max);
    ComputeType thread_sum = 0;
    for (int col = tid; col < cols; col += block_size) {
      if (algorithm == Algorithm::kSoftmax) {
        const ComputeType exp_x = Exp(buf[col] - row_max);
        buf[col] = exp_x;
        thread_sum += exp_x;
      } else {
        const ComputeType x = buf[col] - row_max;
        buf[col] = x;
        thread_sum += Exp(x);
      }
    }
    // 同理，获得当前行的sum
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    // 计算结果并写回到全局内存中
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = Div(buf[i * num_packs + pack_id], row_sum);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = buf[i * num_packs + pack_id] - Log(row_sum);
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxBlockSMemImpl(hipStream_t stream, LOAD load, STORE store, int smem,
                                              const int64_t rows, const int64_t cols) {
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, smem, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

// 执行的主体循环逻辑如下，根据 num_cols算出需要的 Shared Memory 大小作为 Launch Kernel 参数，
// 借助 Shared Memory 保存输入，后续计算直接从 Shared Memory 读取。
// 由于 SM 内的 Shared Memory 资源同样有限，因此当 num_cols超过一定范围，kernel 启动时申请 Shared Memory 超过最大限制，
// 就会出现无法启动的问题，因此，仅在调用 hipOccupancyMaxActiveBlocksPerMultiprocessor 返回值大于0时采用 Shared Memory 方案。
// 此外，需要注意的是，由于 Block 内线程要做同步，当 SM 中正在调度执行的一个 Block 到达同步点时，SM 内可执行 Warp 逐渐减少，
// 若同时执行的 Block 只有一个，则 SM 中可同时执行的 Warp 会在此时逐渐降成0，会导致计算资源空闲，造成浪费，若此时同时有其他 Block 在执行，
// 则在一个 Block 到达同步点时仍然有其他 Block 可以执行。当 block_size 越小时，SM 可同时调度的 Block 越多，因此在这种情况下 block_size 越小越好。
// 但是当在调大 block_size，SM 能同时调度的 Block 数不变的情况下，block_size 应该是越大越好，越大就有越好的并行度。
// 因此代码中在选择 block_size 时，对不同 block_size 都计算了 hipOccupancyMaxActiveBlocksPerMultiprocessor，若结果相同，使用较大的 block_size。
template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxBlockSMemImplBlockSize(hipStream_t stream, LOAD load,
                                                            STORE store, const int64_t rows,
                                                            const int64_t cols, bool* success) {
  // 设置4个不同的block_size
  constexpr int block_size_conf_1 = 128;
  constexpr int block_size_conf_2 = 256;
  constexpr int block_size_conf_3 = 512;
  constexpr int block_size_conf_4 = 1024;
  // 计算第二种方案需要的共享内存大小
  const size_t smem = cols * sizeof(ComputeType);
  int max_active_blocks_conf_1;
  {
    // 占用计算器API hipOccupancyMaxActiveBlocksPerMultiprocessor可以根据 kernel 的 block 大小和共享内存使用情况提供占用率预测。
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_1,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1, algorithm>,
        block_size_conf_1, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_1 <= 0) {
    *success = false;
    return hipSuccess;
  }
  int max_active_blocks_conf_4;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_4,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4, algorithm>,
        block_size_conf_4, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_4 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  int max_active_blocks_conf_3;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_3,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3, algorithm>,
        block_size_conf_3, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_3 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  int max_active_blocks_conf_2;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_2,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2, algorithm>,
        block_size_conf_2, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_2 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  *success = true;
  return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1,
                                    algorithm>(stream, load, store, smem, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct TryDispatchSoftmaxBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, bool* success) {
    if (cols % 2 == 0) {
      return TryDispatchSoftmaxBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 2, algorithm>(
          stream, load, store, rows, cols, success);
    } else {
      return TryDispatchSoftmaxBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 1, algorithm>(
          stream, load, store, rows, cols, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                   const int64_t rows, const int64_t cols,
                                                   bool* success) {
  return TryDispatchSoftmaxBlockSMemImplPackSize<LOAD, STORE, ComputeType, algorithm>()(
      stream, load, store, rows, cols, success);
}

// 和实现2一样，仍然是一个 Block 处理一行元素，
// 不同的是，不再用 Shared Memory 缓存输入x，
// 而是在每次计算时重新读输入 x，
// 这种实现没有最大 num_cols的限制，可以支持任意大小。
// 此外，需要注意的是，在这种实现中，block_size 应该设越大越好，
// block_size 越大，SM 中能同时并行执行的 Block 数就越少，
// 对 cache 的需求就越少，就有更多机会命中 Cache，
// 多次读x不会多次访问 Global Memory，因此在实际测试中，
// 在能利用 Cache 情况下，有效带宽不会因为读3次x而降低几倍。
template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
__global__ void SoftmaxBlockUncachedImpl(LOAD load, STORE store, const int64_t rows,
                                         const int64_t cols) {
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_max = -Inf<ComputeType>();
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { thread_max = max(thread_max, pack[i]); }
    }
    const ComputeType row_max = BlockAllReduce<MaxOp, ComputeType, block_size>(thread_max);
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { thread_sum += Exp(pack[i] - row_max); }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = Div(Exp(pack[i] - row_max), row_sum);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = (pack[i] - row_max) - Log(row_sum);
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
inline hipError_t LaunchSoftmaxBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                  const int64_t rows, const int64_t cols) {
  // 每个 Block 使用 1024 个线程
  constexpr int block_size = 1024;
  // waves 需要满足32组
  constexpr int waves = 32;
  // 根据 BlockSize 以及硬件参数计算 Block 数量
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  // 启动第三者实现的 cuda kernel
  SoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, 0, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct DispatchSoftmaxBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols) {
    // 如果 cols % 2 == 0，就执行 pack2，否则不 pack
    if (cols % 2 == 0) {
      return LaunchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, 2, algorithm>(
          stream, load, store, rows, cols);
    } else {
      return LaunchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, 1, algorithm>(
          stream, load, store, rows, cols);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t DispatchSoftmaxBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                    const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImplPackSize<LOAD, STORE, ComputeType, algorithm>()(
      stream, load, store, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                const int64_t cols) {
  if (cols < 1024) {
    return DispatchSoftmaxWarpImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
        stream, load, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
              stream, load, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
          stream, load, store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
      stream, load, store, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                   const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxWarpImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
        stream, load, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
              stream, load, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
          stream, load, store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                   const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
      stream, load, store, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, bool padding,
         Algorithm algorithm>
__global__ void SoftmaxGradWarpImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
                                    const int64_t cols) {
  static_assert(cols_per_thread % pack_size == 0, "");
  constexpr int pack_per_thread = cols_per_thread / pack_size;
  assert(cols <= cols_per_thread * thread_group_width);
  static_assert(thread_group_width <= kWarpSize, "");
  static_assert(kWarpSize % thread_group_width == 0, "");
  ComputeType y_buf[rows_per_access][cols_per_thread];
  ComputeType dy_buf[rows_per_access][cols_per_thread];
  const int global_thread_group_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int num_global_thread_group = gridDim.x * blockDim.y;
  const int lane_id = threadIdx.x;
  const int64_t step = num_global_thread_group * rows_per_access;
  for (int64_t row = global_thread_group_id * rows_per_access; row < rows; row += step) {
    ComputeType thread_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_sum[row_id] = 0;
      ComputeType* row_y_buf = y_buf[row_id];
      ComputeType* row_dy_buf = dy_buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < pack_per_thread; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          load_y.template load<pack_size>(row_y_buf + pack_offset, row + row_id, col);
          load_dy.template load<pack_size>(row_dy_buf + pack_offset, row + row_id, col);
#pragma unroll
          for (int i = 0; i < pack_size; ++i) {
            if (algorithm == Algorithm::kSoftmax) {
              thread_sum[row_id] += row_y_buf[pack_offset + i] * row_dy_buf[pack_offset + i];
            } else if (algorithm == Algorithm::kLogSoftmax) {
              thread_sum[row_id] += row_dy_buf[pack_offset + i];
            } else {
              __trap();
            }
          }
        }
      }
    }
    ComputeType warp_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_sum[row_id] = WarpAllReduce<SumOp, ComputeType, thread_group_width>(thread_sum[row_id]);
    }
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      ComputeType* row_y_buf = y_buf[row_id];
      ComputeType* row_dy_buf = dy_buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < pack_per_thread; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          for (int i = 0; i < pack_size; ++i) {
            if (algorithm == Algorithm::kSoftmax) {
              row_dy_buf[pack_offset + i] =
                  (row_dy_buf[pack_offset + i] - warp_sum[row_id]) * row_y_buf[pack_offset + i];
            } else if (algorithm == Algorithm::kLogSoftmax) {
              row_dy_buf[pack_offset + i] -= Exp(row_y_buf[pack_offset + i]) * warp_sum[row_id];
            } else {
              __trap();
            }
          }
          store.template store<pack_size>(row_dy_buf + pack_offset, row + row_id, col);
        }
      }
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, bool padding,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradWarpImpl(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy,
                                             STORE store, const int64_t rows, const int64_t cols) {
  constexpr int block_size = 128;
  constexpr int waves = 32;
  static_assert(block_size % thread_group_width == 0, "");
  constexpr int thread_groups_per_block = block_size / thread_group_width;
  dim3 block_dim(thread_group_width, thread_groups_per_block);
  const int64_t num_blocks =
      (rows / rows_per_access + thread_groups_per_block - 1) / thread_groups_per_block;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, num_blocks, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, cols_per_thread,
                      thread_group_width, rows_per_access, padding, algorithm>
      <<<grid_dim_x, block_dim, 0, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradWarpImplPadding(hipStream_t stream, LOAD_Y load_y,
                                                      LOAD_DY load_dy, STORE store,
                                                      const int64_t rows, const int64_t cols) {
  if (cols == cols_per_thread * thread_group_width) {
    return LaunchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                     cols_per_thread, thread_group_width, rows_per_access, false,
                                     algorithm>(stream, load_y, load_dy, store, rows, cols);
  } else {
    return LaunchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                     cols_per_thread, thread_group_width, rows_per_access, true,
                                     algorithm>(stream, load_y, load_dy, store, rows, cols);
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
typename std::enable_if<pack_size == 1, hipError_t>::type DispatchSoftmaxGradWarpImplCols(
    hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
    const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                     \
  else if (cols <= (thread_group_width)*pack_size) {                                            \
    if (rows % 2 == 0) {                                                                        \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 2, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    } else {                                                                                    \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 1, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    }                                                                                           \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                       \
  else if (cols <= (col)*kWarpSize) {                                                              \
    return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, col, \
                                              kWarpSize, 1, algorithm>(stream, load_y, load_dy,    \
                                                                       store, rows, cols);         \
  }
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(3)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(5)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(7)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(9)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(11)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(13)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(15)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(17)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(19)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(21)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(23)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(25)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(27)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(29)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(31)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
typename std::enable_if<pack_size == 2, hipError_t>::type DispatchSoftmaxGradWarpImplCols(
    hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
    const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                     \
  else if (cols <= (thread_group_width)*pack_size) {                                            \
    if (rows % 2 == 0) {                                                                        \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 2, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    } else {                                                                                    \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 1, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    }                                                                                           \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                       \
  else if (cols <= (col)*kWarpSize) {                                                              \
    return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, col, \
                                              kWarpSize, 1, algorithm>(stream, load_y, load_dy,    \
                                                                       store, rows, cols);         \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct DispatchSoftmaxGradWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols) {
    if (cols % 2 == 0) {
      return DispatchSoftmaxGradWarpImplCols<LOAD_Y, LOAD_DY, STORE, ComputeType, 2, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    } else {
      return DispatchSoftmaxGradWarpImplCols<LOAD_Y, LOAD_DY, STORE, ComputeType, 1, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradWarpImpl(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy,
                                               STORE store, const int64_t rows,
                                               const int64_t cols) {
  return DispatchSoftmaxGradWarpImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType, algorithm>()(
      stream, load_y, load_dy, store, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
__global__ void SoftmaxGradBlockSMemImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                                         const int64_t rows, const int64_t cols) {
  extern __shared__ __align__(sizeof(double)) unsigned char grad_shared_buf[];
  auto* y_buf = reinterpret_cast<ComputeType*>(grad_shared_buf);
  auto* dy_buf = y_buf + cols;
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        y_buf[i * num_packs + pack_id] = y_pack[i];
        dy_buf[i * num_packs + pack_id] = dy_pack[i];
        if (algorithm == Algorithm::kSoftmax) {
          thread_sum += y_pack[i] * dy_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          thread_sum += dy_pack[i];
        } else {
          __trap();
        }
      }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = (dy_buf[i * num_packs + pack_id] - row_sum) * y_buf[i * num_packs + pack_id];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = dy_buf[i * num_packs + pack_id] - Exp(y_buf[i * num_packs + pack_id]) * row_sum;
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradBlockSMemImpl(hipStream_t stream, LOAD_Y load_y,
                                                  LOAD_DY load_dy, STORE store, int smem,
                                                  const int64_t rows, const int64_t cols) {
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, smem, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxGradBlockSMemImplBlockSize(hipStream_t stream, LOAD_Y load_y,
                                                                LOAD_DY load_dy, STORE store,
                                                                const int64_t rows,
                                                                const int64_t cols, bool* success) {
  constexpr int block_size_conf_1 = 128;
  constexpr int block_size_conf_2 = 256;
  constexpr int block_size_conf_3 = 512;
  constexpr int block_size_conf_4 = 1024;
  const size_t smem = cols * sizeof(ComputeType) * 2;
  int max_active_blocks_conf_1;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_1,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_1,
                                 algorithm>,
        block_size_conf_1, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_1 <= 0) {
    *success = false;
    return hipSuccess;
  }
  int max_active_blocks_conf_4;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_4,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_4,
                                 algorithm>,
        block_size_conf_4, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_4 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_4, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  int max_active_blocks_conf_3;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_3,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_3,
                                 algorithm>,
        block_size_conf_3, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_3 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_3, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  int max_active_blocks_conf_2;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_2,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_2,
                                 algorithm>,
        block_size_conf_2, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_2 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_2, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  *success = true;
  return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                        block_size_conf_1, algorithm>(stream, load_y, load_dy,
                                                                      store, smem, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct TryDispatchSoftmaxGradBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols, bool* success) {
    if (cols % 2 == 0) {
      return TryDispatchSoftmaxGradBlockSMemImplBlockSize<LOAD_Y, LOAD_DY, STORE, ComputeType, 2,
                                                          algorithm>(stream, load_y, load_dy, store,
                                                                     rows, cols, success);
    } else {
      return TryDispatchSoftmaxGradBlockSMemImplBlockSize<LOAD_Y, LOAD_DY, STORE, ComputeType, 1,
                                                          algorithm>(stream, load_y, load_dy, store,
                                                                     rows, cols, success);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxGradBlockSMemImpl(hipStream_t stream, LOAD_Y load_y,
                                                       LOAD_DY load_dy, STORE store,
                                                       const int64_t rows, const int64_t cols,
                                                       bool* success) {
  return TryDispatchSoftmaxGradBlockSMemImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                     algorithm>()(stream, load_y, load_dy, store,
                                                                  rows, cols, success);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
__global__ void SoftmaxGradBlockUncachedImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                                             const int64_t rows, const int64_t cols) {
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);

#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          thread_sum += y_pack[i] * dy_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          thread_sum += dy_pack[i];
        } else {
          __trap();
        }
      }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          dy_pack[i] = (dy_pack[i] - row_sum) * y_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          dy_pack[i] -= Exp(y_pack[i]) * row_sum;
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(dy_pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradBlockUncachedImpl(hipStream_t stream, LOAD_Y load_y,
                                                      LOAD_DY load_dy, STORE store,
                                                      const int64_t rows, const int64_t cols) {
  constexpr int block_size = 1024;
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size,
                               algorithm>
      <<<grid_dim_x, block_size, 0, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct DispatchSoftmaxGradBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols) {
    if (cols % 2 == 0 && cols > kWarpSize) {
      return LaunchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, 2, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    } else {
      return LaunchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, 1, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradBlockUncachedImpl(hipStream_t stream, LOAD_Y load_y,
                                                        LOAD_DY load_dy, STORE store,
                                                        const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                      algorithm>()(stream, load_y, load_dy, store,
                                                                   rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                    const int64_t rows, const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, Algorithm::kSoftmax>(
        stream, load_y, load_dy, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                            Algorithm::kSoftmax>(
          stream, load_y, load_dy, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                  Algorithm::kSoftmax>(stream, load_y, load_dy,
                                                                       store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                    const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                              Algorithm::kSoftmax>(stream, load_y, load_dy, store,
                                                                   rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                       const int64_t rows, const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, Algorithm::kLogSoftmax>(
        stream, load_y, load_dy, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                            Algorithm::kLogSoftmax>(
          stream, load_y, load_dy, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                  Algorithm::kLogSoftmax>(stream, load_y, load_dy,
                                                                          store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                       const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                              Algorithm::kLogSoftmax>(stream, load_y, load_dy,
                                                                      store, rows, cols);
}

int main(){
  const int rows = 32 * 64 * 512;
  const int cols = 512;
  const int N = rows * cols;
  using ComputeType = typename DefaultComputeType<float>::type;
  float* input_host = (float*)malloc(N*sizeof(float));
  float *input_device;
  hipMalloc((void **)&input_device, N*sizeof(float));
  for (int i = 0; i < N; i++) input_host[i] = 1.0;
  hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);
  DirectLoad<float, ComputeType> load(input_device, cols);

  float *output_host = (float*)malloc(N * sizeof(float));
  float *output_device;
  hipMalloc((void **)&output_device, N * sizeof(float));
  DirectStore<ComputeType, float> store(output_device, cols);
  
  hipStream_t stream;
  hipStreamCreate(&stream);
  DispatchSoftmax<decltype(load), decltype(store), ComputeType>(
        stream, load, store, rows, cols);
  CUDA_CHECK();
  hipMemcpy(output_host, output_device, N * sizeof(float), hipMemcpyDeviceToHost);
  // 1 / 32 = 0.03125
  for (int i = 0; i < 32; i++){
    printf("%.5f\n", output_host[i]);
  }
  hipFree(input_device);
  hipFree(output_device);
  free(input_host);
  free(output_host);
  return 0;
}