#include <hip/hip_runtime.h>

#include <time.h>
#include <algorithm>
#include <iostream>
#include <hip/hip_fp16.h>
using namespace std;

#define N 32 * 1024 * 1024
// elementwise implementation copyed from https://github.com/Oneflow-Inc/oneflow/blob/master/oneflow/core/cuda/elementwise.cuh
constexpr int kBlockSize = 256;
constexpr int kNumWaves = 32;

inline hipError_t GetNumBlocks(int64_t n, int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks = std::max<int>(1, std::min<int64_t>((n + kBlockSize - 1) / kBlockSize,
                                                   sm_count * tpm / kBlockSize * kNumWaves));
  return hipSuccess;
}

template<typename T, int pack_size>
struct GetPackType {
  using type = typename std::aligned_storage<pack_size * sizeof(T), pack_size * sizeof(T)>::type;
};

template<typename T, int pack_size>
using PackType = typename GetPackType<T, pack_size>::type;

template<typename T, int pack_size>
union Pack {
  static_assert(sizeof(PackType<T, pack_size>) == sizeof(T) * pack_size, "");
  __device__ Pack() {
    // do nothing
  }
  PackType<T, pack_size> storage;
  T elem[pack_size];
};

template<typename T, int pack_size>
struct alignas(sizeof(T) * pack_size) Packed {
  __device__ Packed() {
    // do nothing
  }
  union {
    T elem[pack_size];
  };
};

constexpr int kMaxPackBytes = 128 / 8;
constexpr int kMaxPackSize = 8;

constexpr int Min(int a, int b) { return a < b ? a : b; }

template<typename T>
constexpr int PackSize() {
  return Min(kMaxPackBytes / sizeof(T), kMaxPackSize);
}

template<typename T, typename U, typename... Args>
constexpr int PackSize() {
  return Min(PackSize<T>(), PackSize<U, Args...>());
}

template<typename T>
class HasApply2 {
  typedef char one;
  struct two {
    char x[2];
  };

  template<typename C>
  static one test(decltype(&C::Apply2));
  template<typename C>
  static two test(...);

 public:
  enum { value = sizeof(test<T>(0)) == sizeof(char) };
};

template<int pack_size, typename FunctorT, typename R, typename... IN>
__device__ typename std::enable_if<HasApply2<FunctorT>::value == true && pack_size % 2 == 0,
                                   Packed<R, pack_size>>::type
ApplyPack(const FunctorT& functor, const Packed<IN, pack_size>... in) {
  Packed<R, pack_size> ret;
#pragma unroll
  for (int j = 0; j < pack_size; j += 2) { functor.Apply2(ret.elem + j, (in.elem + j)...); }
  return ret;
}

template<int pack_size, typename FunctorT, typename R, typename... IN>
__device__ typename std::enable_if<HasApply2<FunctorT>::value == false || pack_size % 2 != 0,
                                   Packed<R, pack_size>>::type
ApplyPack(const FunctorT& functor, const Packed<IN, pack_size>... in) {
  Packed<R, pack_size> ret;
#pragma unroll
  for (int j = 0; j < pack_size; ++j) { ret.elem[j] = functor((in.elem[j])...); }
  return ret;
}

template<int pack_size, typename FactoryT, typename R, typename... IN>
__global__ void __launch_bounds__(kBlockSize)
    ApplyGeneric(FactoryT factory, int64_t n_pack, Packed<R, pack_size>* pack_r,
                 const Packed<IN, pack_size>*... pack_in, int64_t n_tail, R* tail_r,
                 const IN*... tail_in) {
  auto functor = factory();
  const int global_tid = blockIdx.x * kBlockSize + threadIdx.x;
  for (int64_t i = global_tid; i < n_pack; i += blockDim.x * gridDim.x) {
    pack_r[i] = ApplyPack<pack_size, decltype(functor), R, IN...>(functor, (pack_in[i])...);
  }
  if (global_tid < n_tail) { tail_r[global_tid] = functor((tail_in[global_tid])...); }
}

template<typename FunctorT>
struct SimpleFactory {
  explicit SimpleFactory(FunctorT functor) : tpl(functor) {}
  __device__ FunctorT operator()() const { return tpl; }

 private:
  FunctorT tpl;
};

template<size_t pack_size>
bool IsAlignedForPack() {
  return true;
}

template<size_t pack_size, typename T, typename... Args>
bool IsAlignedForPack(const T* ptr, const Args*... others) {
  return reinterpret_cast<uintptr_t>(ptr) % sizeof(Pack<T, pack_size>) == 0
         && IsAlignedForPack<pack_size, Args...>(others...);
}

template<size_t pack_size, typename FactoryT, typename R, typename... IN>
hipError_t LaunchKernel(FactoryT factory, int64_t n, R* r, const IN*... in) {
  const int64_t n_pack = n / pack_size;
  const int64_t tail_offset = n_pack * pack_size;
  const int64_t n_tail = n - tail_offset;
  int num_blocks;
  {
    hipError_t err = GetNumBlocks(n_pack, &num_blocks);
    if (err != hipSuccess) { return err; }
  }
  ApplyGeneric<pack_size, FactoryT, R, IN...><<<num_blocks, kBlockSize, 0>>>(
      factory, n_pack, reinterpret_cast<Packed<R, pack_size>*>(r),
      (reinterpret_cast<const Packed<IN, pack_size>*>(in))..., n_tail, r + tail_offset,
      (in + tail_offset)...);
  return hipPeekAtLastError();
}

template<typename FactoryT, typename R, typename... IN>
struct GenericLauncher {
  static hipError_t Launch(FactoryT factory, int64_t n, R* r, const IN*... in) {
    constexpr int max_pack_size = PackSize<R, IN...>();
    if (IsAlignedForPack<max_pack_size, R, IN...>(r, in...)) {
      return LaunchKernel<max_pack_size, FactoryT, R, IN...>(factory, n, r, in...);
    } else {
      return LaunchKernel<1, FactoryT, R, IN...>(factory, n, r, in...);
    }
  }
};

template<typename FactoryT, typename R, typename A>
inline hipError_t UnaryWithFactory(FactoryT factory, int64_t n, R* r, const A* a) {
  return GenericLauncher<FactoryT, R, A>::Launch(factory, n, r, a);
}

template<typename FunctorT, typename R, typename A>
inline hipError_t Unary(FunctorT functor, int64_t n, R* r, const A* a) {
  return UnaryWithFactory(SimpleFactory<FunctorT>(functor), n, r, a);
}

template<typename FactoryT, typename R, typename A, typename B>
inline hipError_t BinaryWithFactory(FactoryT factory, int64_t n, R* r, const A* a, const B* b) {
  return GenericLauncher<FactoryT, R, A, B>::Launch(factory, n, r, a, b);
}

template<typename FunctorT, typename R, typename A, typename B>
inline hipError_t Binary(FunctorT functor, int64_t n, R* r, const A* a, const B* b) {
  return BinaryWithFactory(SimpleFactory<FunctorT>(functor), n, r, a, b);
}

template<typename FactoryT, typename R, typename A, typename B, typename C>
inline hipError_t TernaryWithFactory(FactoryT factory, int64_t n, R* r, const A* a, const B* b,
                                      const C* c) {
  return GenericLauncher<FactoryT, R, A, B, C>::Launch(factory, n, r, a, b, c);
}

template<typename FunctorT, typename R, typename A, typename B, typename C>
inline hipError_t Ternary(FunctorT functor, int64_t n, R* r, const A* a, const B* b, const C* c) {
  return TernaryWithFactory(SimpleFactory<FunctorT>(functor), n, r, a, b, c);
}

template<typename T>
struct MultiplyFunctor {
  __device__ T operator()(T x, T y) const {
    return x*y;
  }
};

template<>
struct MultiplyFunctor<half> {
  __device__ half operator()(half x, half y) const {
    return x*y;
  }
#if (__CUDA_ARCH__ >= 750 && CUDA_VERSION >= 11000)
  __device__ void Apply2(half* z, const half* x, const half* y) const {
    const half2 x2 = *(reinterpret_cast<const half2*>(x));
    const half2 y2 = *(reinterpret_cast<const half2*>(y));
    *reinterpret_cast<half2*>(z) = __hmul2(x2, y2);
  }
#endif  // (__CUDA_ARCH__ >= 750 && CUDA_VERSION >= 11000)
};

template<typename T>
__global__ void mul(T *x, T *y, T* z){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  z[idx] = x[idx] * y[idx];
}

template<>
__global__ void mul(half *x, half *y, half* z){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  z[idx] = x[idx] * y[idx];
}



int main(){
    half *x_host = (half*)malloc(N*sizeof(half));
    half *x_device;
    hipMalloc((void **)&x_device, N*sizeof(half));
    for (int i = 0; i < N; i++) x_host[i] = 2.0;
    hipMemcpy(x_device, x_host, N*sizeof(half), hipMemcpyHostToDevice);

    half *y_host = (half*)malloc(N*sizeof(half));
    half *y_device;
    hipMalloc((void **)&y_device, N*sizeof(half));
    for (int i = 0; i < N; i++) y_host[i] = 2.0;
    hipMemcpy(y_device, y_host, N*sizeof(half), hipMemcpyHostToDevice);

    half *output_host = (half*)malloc(N * sizeof(half));
    half *output_device;
    hipMalloc((void **)&output_device, N * sizeof(half));

    // naive elementwise
    int32_t block_num = (N + kBlockSize - 1) / kBlockSize;
    dim3 grid(block_num, 1);
    dim3 block(kBlockSize, 1);
    mul<half><<<grid, block>>>(x_device, y_device, output_device);
    hipMemcpy(output_host, output_device, N * sizeof(half), hipMemcpyDeviceToHost);

    // elementwise template
    Binary(MultiplyFunctor<half>(), N, output_device, x_device, y_device);
    hipMemcpy(output_host, output_device, N * sizeof(half), hipMemcpyDeviceToHost);
    free(x_host);
    free(y_host);
    free(output_host);
    hipFree(x_device);
    hipFree(y_device);
    hipFree(output_device);
    return 0;
}

// float dtype
// int main(){
//     float *x_host = (float*)malloc(N*sizeof(float));
//     float *x_device;
//     cudaMalloc((void **)&x_device, N*sizeof(float));
//     for (int i = 0; i < N; i++) x_host[i] = 2.0;
//     cudaMemcpy(x_device, x_host, N*sizeof(float), cudaMemcpyHostToDevice);

//     float *y_host = (float*)malloc(N*sizeof(float));
//     float *y_device;
//     cudaMalloc((void **)&y_device, N*sizeof(float));
//     for (int i = 0; i < N; i++) y_host[i] = 2.0;
//     cudaMemcpy(y_device, y_host, N*sizeof(float), cudaMemcpyHostToDevice);

//     float *output_host = (float*)malloc(N * sizeof(float));
//     float *output_device;
//     cudaMalloc((void **)&output_device, N * sizeof(float));

//     // naive elementwise
//     int32_t block_num = (N + kBlockSize - 1) / kBlockSize;
//     dim3 grid(block_num, 1);
//     dim3 block(kBlockSize, 1);
//     mul<float><<<grid, block>>>(x_device, y_device, output_device);
//     cudaMemcpy(output_host, output_device, N * sizeof(float), cudaMemcpyDeviceToHost);

//     // elementwise template
//     Binary(MultiplyFunctor<float>(), N, output_device, x_device, y_device);
//     cudaMemcpy(output_host, output_device, N * sizeof(float), cudaMemcpyDeviceToHost);
//     free(x_host);
//     free(y_host);
//     free(output_host);
//     cudaFree(x_device);
//     cudaFree(y_device);
//     cudaFree(output_device);
//     return 0;
// }
