#include <hip/hip_runtime.h>

#include <time.h>

#define N 32*1024*1024
#define BLOCK_SIZE 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache,int tid){
    if(blockSize >= 64)cache[tid]+=cache[tid+32];
    if(blockSize >= 32)cache[tid]+=cache[tid+16];
    if(blockSize >= 16)cache[tid]+=cache[tid+8];
    if(blockSize >= 8)cache[tid]+=cache[tid+4];
    if(blockSize >= 4)cache[tid]+=cache[tid+2];
    if(blockSize >= 2)cache[tid]+=cache[tid+1];
}

template <unsigned int blockSize>
__global__ void reduce_v5(float *g_idata,float *g_odata){
    __shared__ float sdata[BLOCK_SIZE];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    if(blockSize>=512){
        if(tid<256){
            sdata[tid]+=sdata[tid+256];
        }
        __syncthreads();
    }
    if(blockSize>=256){
        if(tid<128){
            sdata[tid]+=sdata[tid+128];
        }
        __syncthreads();
    }
    if(blockSize>=128){
        if(tid<64){
            sdata[tid]+=sdata[tid+64];
        }
        __syncthreads();
    }
    
    // write result for this block to global mem
    if(tid<32)warpReduce<blockSize>(sdata,tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    float *input_host = (float*)malloc(N*sizeof(float));
    float *input_device;
    hipMalloc((void **)&input_device, N*sizeof(float));
    for (int i = 0; i < N; i++) input_host[i] = 2.0;
    hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);

    int32_t block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE / 2;
    float *output_host = (float*)malloc((block_num) * sizeof(float));
    float *output_device;
    hipMalloc((void **)&output_device, (block_num) * sizeof(float));
    
    dim3 grid(block_num, 1);
    dim3 block(BLOCK_SIZE, 1);
    reduce_v5<BLOCK_SIZE><<<grid, block>>>(input_device, output_device);
    hipMemcpy(output_host, output_device, block_num * sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}


