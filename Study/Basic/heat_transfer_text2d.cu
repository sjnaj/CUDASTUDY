#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "parallelprogrammingbook/include/hpc_helpers.hpp"
#include "common/cpu_bitmap.h"
#include "common/cpu_anim.h"
#include "common/book.h"
#define DIM 1024
#define MAX_TEMP 0.8f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

//纹理引用被弃用，效果不如访问global变量 

// 需要全局声明，不能作为函数参数传递
texture<float, 2> texConstSrc;
texture<float, 2> texIn;
texture<float, 2> texOut;

/*heating cells remains at a constant temperature*/
__global__ void copy_const_kernel(float *iptr,hipTextureObject_t const constSrc)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex2D<float>(constSrc, x, y);
    if (c != 0)
    {
        iptr[offset] = c;
    }
}

__global__ void blend_kernel(float *dst, bool dstOut)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float t, l, c, r, b;

    if (dstOut)
    {
        // 越界的会自动回退
        t = tex2D(texIn, x, y - 1);
        l = tex2D(texIn, x - 1, y);
        c = tex2D(texIn, x, y);
        r = tex2D(texIn, x + 1, y);
        b = tex2D(texIn, x, y + 1);
    }
    else
    {
        t = tex2D(texOut, x, y - 1);
        l = tex2D(texOut, x - 1, y);
        c = tex2D(texOut, x, y);
        r = tex2D(texOut, x + 1, y);
        b = tex2D(texOut, x, y + 1);
    }
    dst[offset] = c + SPEED * (t + l + r + b - 4 * c); // 通过纹理访问输入，通过global修改输出。
}

struct DataBlock
{
    unsigned char *output_bitmap;
    float *dev_inSrc;
    float *dev_outSrc;
    float *dev_constSrc;
    hipTextureObject_t text_constSrc;
    CPUAnimBitmap *bitmap;
    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

void anim_gpu(DataBlock *d, int ticks)
{
    hipEventRecord(d->start, 0);
    CUERR
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;

    volatile bool dstOut = true;

    for (int i = 0; i < 90; i++)
    {
        float *in, *out;
        if (dstOut)
        {
            in = d->dev_inSrc;
            out = d->dev_outSrc;
        }
        else
        {
            out = d->dev_inSrc;
            in = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks, threads>>>(in,d->text_constSrc);
        blend_kernel<<<blocks, threads>>>(out, dstOut);
        dstOut = !dstOut;
    }

    float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);

    hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
    CUERR
    hipEventRecord(d->stop, 0);
    CUERR
    hipEventSynchronize(d->stop);
    CUERR
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, d->start, d->stop);
    CUERR
    d->totalTime += elapsedTime;
    d->frames++;
    printf("average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d)
{
    hipUnbindTexture(texIn);
    CUERR
    hipUnbindTexture(texOut);
    CUERR
    hipUnbindTexture(texConstSrc);
    CUERR
    hipFree(d->dev_inSrc);
    CUERR
    hipFree(d->dev_outSrc);
    CUERR
    hipFree(d->dev_constSrc);
    CUERR
    hipEventDestroy(d->start);
    CUERR
    hipEventDestroy(d->stop);
    CUERR
}

int main(void)
{

    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    long const imageSize = bitmap.image_size();
    hipEventCreate(&data.start);
    CUERR
    hipEventCreate(&data.stop);
    CUERR
    hipMalloc((void **)&data.output_bitmap, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_inSrc, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_outSrc, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_constSrc, imageSize);
    CUERR

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(NULL, texConstSrc, data.dev_constSrc, desc, DIM, DIM, sizeof(float) * DIM);
    CUERR
    hipBindTexture2D(NULL, texIn, data.dev_inSrc, desc, DIM, DIM, sizeof(float) * DIM);
    CUERR
    hipBindTexture2D(NULL, texOut, data.dev_outSrc, desc, DIM, DIM, sizeof(float) * DIM);
    CUERR

    float *temp;
    hipHostMalloc((void **)&temp, imageSize);
    CUERR

    // 初始化内部热点
    for (int i = 0; i < DIM * DIM; i++)
    {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
        {
            temp[i] = MAX_TEMP;
        }
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++)
    {
        for (int x = 400; x < 500; x++)
        {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }

    // hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipArray *cuArray;
    hipMallocArray(&cuArray, &desc, DIM * sizeof(float), DIM);
    hipMemcpy2DToArray(cuArray, 0, 0, temp, DIM * sizeof(float), DIM * sizeof(float), DIM, hipMemcpyHostToDevice);
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;//越界的设为边界值
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipCreateTextureObject(&data.text_constSrc, &resDesc, &texDesc, NULL);

    hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR
    // 初始化外部影响
    for (int y = 800; y < DIM; y++)
    {
        for (int x = 0; x < 200; x++)
        {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR

    hipHostFree(temp);
    CUERR

    bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void *))anim_exit);
}
