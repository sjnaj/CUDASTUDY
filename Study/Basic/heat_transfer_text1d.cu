#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "parallelprogrammingbook/include/hpc_helpers.hpp"
#include "common/cpu_bitmap.h"
#include "common/cpu_anim.h"
#include "common/book.h"
#define DIM 1024
#define MAX_TEMP 0.8f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

// 需要全局声明，不能作为函数参数传递
texture<float> texConstSrc;
texture<float> texIn;
texture<float> texOut;

/*heating cells remains at a constant temperature*/
__global__ void copy_const_kernel(float *iptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex1Dfetch(texConstSrc, offset);
    if (c != 0)
    {
        iptr[offset] = c;
    }
}

__global__ void blend_kernel(float *dst, bool dstOut)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)
        left++;
    if (x == DIM - 1)
        right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)
        top += DIM;
    if (y == DIM - 1)
        bottom -= DIM;

    float t, l, c, r, b;

    if (dstOut)
    {
        t = tex1Dfetch(texIn, top);
        l = tex1Dfetch(texIn, left);
        c = tex1Dfetch(texIn, offset);
        r = tex1Dfetch(texIn, right);
        b = tex1Dfetch(texIn, bottom);
    }
    else
    {
        t = tex1Dfetch(texOut, top);
        l = tex1Dfetch(texOut, left);
        c = tex1Dfetch(texOut, offset);
        r = tex1Dfetch(texOut, right);
        b = tex1Dfetch(texOut, bottom);
    }
    dst[offset] = c + SPEED * (t + l + r + b - 4 * c);//通过纹理访问输入，通过global修改输出。
}

struct DataBlock
{
    unsigned char *output_bitmap;
    float *dev_inSrc;
    float *dev_outSrc;
    float *dev_constSrc;
    CPUAnimBitmap *bitmap;
    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

void anim_gpu(DataBlock *d, int ticks)
{
    hipEventRecord(d->start, 0);
    CUERR
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;

    volatile bool dstOut = true;

    for (int i = 0; i < 90; i++)
    {
        float *in, *out;
        if (dstOut)
        {
            in = d->dev_inSrc;
            out = d->dev_outSrc;
        }
        else
        {
            out = d->dev_inSrc;
            in = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks, threads>>>(in);
        blend_kernel<<<blocks, threads>>>(out, dstOut);
        dstOut = !dstOut;
    }

    float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);

    hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
    CUERR
    hipEventRecord(d->stop, 0);
    CUERR
    hipEventSynchronize(d->stop);
    CUERR
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, d->start, d->stop);
    CUERR
    d->totalTime += elapsedTime;

    d->frames++;
    printf("average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d)
{
    hipUnbindTexture(texIn);
    CUERR
    hipUnbindTexture(texOut);
    CUERR
    hipUnbindTexture(texConstSrc);
    CUERR
    hipFree(d->dev_inSrc);
    CUERR
    hipFree(d->dev_outSrc);
    CUERR
    hipFree(d->dev_constSrc);
    CUERR

    hipEventDestroy(d->start);
    CUERR
    hipEventDestroy(d->stop);
    CUERR
}

int main(void)
{

    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    long const imageSize = bitmap.image_size();
    hipEventCreate(&data.start);
    CUERR
    hipEventCreate(&data.stop);
    CUERR
    hipMalloc((void **)&data.output_bitmap, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_inSrc, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_outSrc, imageSize);
    CUERR
    hipMalloc((void **)&data.dev_constSrc, imageSize);
    CUERR

    hipBindTexture(NULL, texConstSrc, data.dev_constSrc, imageSize);
    CUERR
    hipBindTexture(NULL, texIn, data.dev_inSrc, imageSize);
    CUERR
    hipBindTexture(NULL, texOut, data.dev_outSrc, imageSize);
    CUERR


    float *temp;
    hipHostMalloc((void **)&temp, imageSize);
    CUERR

    // 初始化内部热点
    for (int i = 0; i < DIM * DIM; i++)
    {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
        {
            temp[i] = MAX_TEMP;
        }
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++)
    {
        for (int x = 400; x < 500; x++)
        {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }

    hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR
    // 初始化外部影响
    for (int y = 800; y < DIM; y++)
    {
        for (int x = 0; x < 200; x++)
        {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
    CUERR

    hipHostFree(temp);
    CUERR

    bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void *))anim_exit);
}
